#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int mandelbrot(double real, double imag, int max_iter)
{
    double z_real = 0.0, z_imag = 0.0;
    int iter = 0;
    while (z_real * z_real + z_imag * z_imag <= 4.0 && iter < max_iter)
    {
        double temp = z_real * z_real - z_imag * z_imag + real;
        z_imag = 2.0 * z_real * z_imag + imag;
        z_real = temp;
        iter++;
    }
    return iter;
}
__device__ void valueToRGB(int color, uint8_t &r, uint8_t &g, uint8_t &b)
{

    double h = (color % 360) / 360.0;
    double s = 0.8;
    double v = 1.0;

    if (color <= 0)
    {
        r = g = b = 0;
        return;
    }

    int i = (int)(h * 6);
    double f = h * 6 - i;
    double p = v * (1 - s);
    double q = v * (1 - f * s);
    double t = v * (1 - (1 - f) * s);

    switch (i % 6)
    {
    case 0:
        r = (uint8_t)(v * 255);
        g = (uint8_t)(t * 255);
        b = (uint8_t)(p * 255);
        break;
    case 1:
        r = (uint8_t)(q * 255);
        g = (uint8_t)(v * 255);
        b = (uint8_t)(p * 255);
        break;
    case 2:
        r = (uint8_t)(p * 255);
        g = (uint8_t)(v * 255);
        b = (uint8_t)(t * 255);
        break;
    case 3:
        r = (uint8_t)(p * 255);
        g = (uint8_t)(q * 255);
        b = (uint8_t)(v * 255);
        break;
    case 4:
        r = (uint8_t)(t * 255);
        g = (uint8_t)(p * 255);
        b = (uint8_t)(v * 255);
        break;
    case 5:
        r = (uint8_t)(v * 255);
        g = (uint8_t)(p * 255);
        b = (uint8_t)(q * 255);
        break;
    }
}

__global__ void render(uint8_t *image, double scale, double centerX, double centerY, int WIDTH, int HEIGHT)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT)
        return;

    double real = (x - WIDTH / 2.0) * scale + centerX;
    double imag = (HEIGHT / 2.0 - y) * scale + centerY; // Korrigierte Y-Achse

    const double INITIAL_SCALE_AT_ZOOM_1 = 4.0 / WIDTH; // Dies ist ein konstanter Wert, der den Skalierungsfaktor bei Zoom 1 repräsentiert.

    int MAX_ITER = 256;
    if (scale > 0)
    {

        MAX_ITER += (int)(log(INITIAL_SCALE_AT_ZOOM_1 / scale) * 50.0);

        if (MAX_ITER < 100)
            MAX_ITER = 100;
        if (MAX_ITER > 8192)
            MAX_ITER = 8192;
    }

    int iter = mandelbrot(real, imag, MAX_ITER);
    int idx = 3 * (y * WIDTH + x);

    uint8_t color = 0;

    if (iter < MAX_ITER)
    {
        double normalized_iter = (double)iter / (double)MAX_ITER;
        color = (uint8_t)(sqrt(normalized_iter) * 255.0);
    }

    uint8_t r, g, b;
    valueToRGB(color, r, g, b);

    image[idx + 0] = r; // R
    image[idx + 1] = g; // G
    image[idx + 2] = b; // B
}

int main()
{
    fprintf(stderr, "CUDA Backend started\n");
    fflush(stderr);

    char line[256];
    
    // CUDA Events außerhalb der Schleife initialisieren, um wiederholte Erzeugung zu vermeiden
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Initialisierung von d_image und h_image auf NULL
    // Dies hilft, potenzielle doppelte Freigaben zu vermeiden, wenn die Schleife vorzeitig beendet wird
    uint8_t *d_image = NULL;
    uint8_t *h_image = NULL;
    size_t currentImageSize = 0; // Speichert die aktuelle Größe des zugewiesenen Speichers

    while (fgets(line, sizeof(line), stdin))
    {
        int WIDTH;  // Breite des Bildes
        int HEIGHT; // Höhe des Bildes
        double zoom, centerX, centerY;
        
        if (sscanf(line, "%lf %lf %lf %d %d", &zoom, &centerX, &centerY, &WIDTH, &HEIGHT) != 5)
        {
            fprintf(stderr, "Invalid input: %s", line);
            fflush(stderr);
            continue;
        }
        
        size_t newImageSize = (size_t)WIDTH * HEIGHT * 3;

        // Speicher nur neu zuweisen, wenn die Größe sich ändert
        if (newImageSize != currentImageSize) {
            if (d_image) {
                hipFree(d_image);
                d_image = NULL;
            }
            if (h_image) {
                free(h_image);
                h_image = NULL;
            }
            hipMalloc(&d_image, newImageSize);
            h_image = (uint8_t *)malloc(newImageSize);
            
            if (h_image == NULL) {
                if (d_image) hipFree(d_image);
                hipEventDestroy(start);
                hipEventDestroy(stop);
                return 1; 
            }
            if (hipGetLastError() != hipSuccess) {
                if (h_image) free(h_image);
                hipEventDestroy(start);
                hipEventDestroy(stop);
                return 1;
            }
            currentImageSize = newImageSize;
        }

        int blockSize = 16;

        dim3 block(blockSize, blockSize);
        dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);

        fprintf(stderr, "Received: zoom=%.2f, centerX=%.2f, centerY=%.2f, WIDTH=%d, HEIGHT=%d\n", zoom, centerX, centerY, WIDTH, HEIGHT);
        fflush(stderr);

        double scale = 4.0 / (WIDTH * zoom);

        // Timing START
        hipEventRecord(start);
        
        hipMemset(d_image, 0, newImageSize); 

        render<<<grid, block>>>(d_image, scale, centerX, centerY, WIDTH, HEIGHT);

        hipDeviceSynchronize();

        // Timing STOP
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds, start, stop);

        hipMemcpy(h_image, d_image, newImageSize, hipMemcpyDeviceToHost);

        fwrite(h_image, 1, newImageSize, stdout);
        fflush(stdout);

        fprintf(stderr, "Frame render time: %.3f ms\n", milliseconds);
        fflush(stderr);
    }

    // Ressourcen freigeben, wenn die Schleife beendet ist
    if (d_image) {
        hipFree(d_image);
    }
    if (h_image) {
        free(h_image);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

    fprintf(stderr, "CUDA Backend clean exit\n");
    fflush(stderr);

    return 0;
}
